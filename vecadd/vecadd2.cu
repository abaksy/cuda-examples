/*
CUDA C++ Program to add 2 1-dimensional vectors of length N
1 blocks, N threads in block
*/
#include<iostream>
#include<hip/hip_runtime.h>
#define N 10

__global__ void vecadd(int* a, int* b, int* c)
{
    int idx = threadIdx.x;
    c[idx] = a[idx] + b[idx]; 
}

int main()
{
    /* Set up variables on host*/
    int* a = new int[N];
    int* b = new int[N];
    int* c = new int[N];

    /* Input values on host*/
    unsigned int size = N*sizeof(int);
    for(int i=0; i<N; ++i)
    {
        a[i] = 2*i;
        b[i] = 3*i+1;
    }

    /* Setting up variables on device. i.e. GPU */
    int* d_a, *d_b, *d_c;
    hipMalloc((void**)&d_a, size);
    hipMalloc((void**)&d_b, size);
    hipMalloc((void**)&d_c, size);

    /* Copy data from host to device */
    hipMemcpy(d_a, a, size, hipMemcpyHostToDevice);
    hipMemcpy(d_b, b, size, hipMemcpyHostToDevice);
    
    /* 
    Kernel Launch
    Grid contains 1 block 
    That block has N threads
    Hence index of vector is thread index
    */
    vecadd<<<1, N>>>(d_a, d_b, d_c);
    hipDeviceSynchronize();

    /* Copy result from GPU device to host */
    hipMemcpy(c, d_c, size, hipMemcpyDeviceToHost);
    
    /* Print result */
    for(int i=0; i<N; ++i)
    {
        std::cout<<c[i]<<' ';
    }
    std::cout<<'\n';

    /* Cleanup device and host memory */
    hipFree(d_a);
    hipFree(d_b);
    hipFree(d_c);
    delete a;
    delete b;
    delete c;

    return 0;
}