/*
CUDA C++ Program to add 2 1-dimensional vectors of length N
N blocks, 1 thread per block
*/
#include<iostream>
#include<hip/hip_runtime.h>
#define N 10

__global__ void vecadd(int* a, int* b, int* c)
{
    int idx = blockIdx.x;
    c[idx] = a[idx] + b[idx]; 
}

int main()
{
    /* Set up variables on host*/
    int* a = new int[N];
    int* b = new int[N];
    int* c = new int[N];

    /* Input values on host*/
    unsigned int size = N*sizeof(int);
    for(int i=0; i<N; ++i)
    {
        a[i] = 2*i;
        b[i] = 3*i+1;
    }

    /* Setting up variables on device. i.e. GPU */
    int* d_a, *d_b, *d_c;
    hipMalloc((void**)&d_a, size);
    hipMalloc((void**)&d_b, size);
    hipMalloc((void**)&d_c, size);

    /* Copy data from host to device */
    hipMemcpy(d_a, a, size, hipMemcpyHostToDevice);
    hipMemcpy(d_b, b, size, hipMemcpyHostToDevice);
    
    /* 
    Kernel Launch
    Grid contains N blocks
    Each block has only one thread
    hence index of vector is block index
    */
    vecadd<<<N, 1>>>(d_a, d_b, d_c);
    hipDeviceSynchronize();
    
    /* Copy result from GPU device to host */
    hipMemcpy(c, d_c, size, hipMemcpyDeviceToHost);
    
    /* Print result */
    for(int i=0; i<N; ++i)
    {
        std::cout<<c[i]<<' ';
    }
    std::cout<<'\n';

    /* Cleanup device and host memory */
    hipFree(d_a);
    hipFree(d_b);
    hipFree(d_c);
    delete a;
    delete b;
    delete c;

    return 0;
}