#include "hip/hip_runtime.h"
/*
Matrix multiplication in CUDA C++
Matrix sizes are M*N and P*Q
Using grid of size M*Q blocks with 1 thread in each block
*/
#include"matmul_utils.hpp"

#define M 3
#define N 3
#define P 3
#define Q 3

__global__ void matmulKernel(int* a, int*b, int* c)
{
    int i = blockIdx.x;
    int j = blockIdx.y;
    c[i*Q + j] = 0;
    /*
    Perform inner product between row i of A and column j of B
    */
    for(int k=0; k<N; ++k)
    {
        c[i*Q + j] += (a[i*N + k] * b[k*Q + j]);
    }
}

int main()
{
    int* a = new int[M * N * sizeof(int)];
    int* b = new int[P * Q * sizeof(int)];
    int* c = new int[M * Q * sizeof(int)];
    init_matrix(a, M, N);
    init_matrix(b, P, Q);

    std::cout<<"A =\n";
    display_matrix(a, M, N);
    std::cout<<"B =\n";
    display_matrix(b, P, Q);

    assert(N==P); //Necessary condition for matrix multiplication

    int *d_a, *d_b, *d_c;
    hipMalloc((void**)&d_a, M * N * sizeof(int));
    hipMalloc((void**)&d_b, P * Q * sizeof(int));
    hipMalloc((void**)&d_c, M * Q * sizeof(int));

    hipMemcpy(d_a, a, M * N * sizeof(int), hipMemcpyHostToDevice);
    hipMemcpy(d_b, b, P * Q * sizeof(int), hipMemcpyHostToDevice);
    
    /*
    Each block of the grid takes care of multiplying one row of A with one column of B

    The single thread in the block performs the inner product of the row of A and column of B
    */
    dim3 grid(M, Q);

    matmulKernel<<<grid, 1>>>(d_a, d_b, d_c);

    hipMemcpy(c, d_c, M * Q * sizeof(int), hipMemcpyDeviceToHost);
    std::cout<<"A*B =\n";
    display_matrix(c, M, Q);

    delete a;
    delete b;
    delete c;

    hipFree(d_a);
    hipFree(d_b);
    hipFree(d_c);
    return 0;
}