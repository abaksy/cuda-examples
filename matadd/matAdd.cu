/*
Program to add 2 matrics of size M * N in CUDA C++
Using 2-D grid of M*N size (i.e. grid contains M*N blocks arranged in 2D fashion)
Each block contains 1 thread
*/
#include<iostream>
#include<hip/hip_runtime.h>
#define M 7
#define N 3

__global__ void matAdd(int* a, int* b, int* c)
{
    int idx = blockIdx.x * gridDim.y + blockIdx.y;
    c[idx] = a[idx] + b[idx];
}

__host__ void print_matrix(int* matrix)
{
    for(int i=0; i<M; ++i)
    {
        for(int j=0; j<N; ++j)
        {
            std::cout<<matrix[i*N+j]<<' ';
        }
        std::cout<<"\n";
    }
    std::cout<<"\n";
}

int main()
{
    int size = M * N * sizeof(int);
    int* a = new int[size];
    int* b = new int[size];
    int* c = new int[size];

    for(int i=0; i<M; ++i)
    {
        for(int j=0; j<N; ++j)
        {
            a[i*N + j] = i; //Fill your own values here
            b[i*N + j] = j; //Fill your own values here
        }
    }
    std::cout<<"MATRIX A =\n";
    print_matrix(a);
    std::cout<<"MATRIX B =\n";
    print_matrix(b);

    /* Setting up variables on device. i.e. GPU */
    int *d_a, *d_b, *d_c;
    hipMalloc((void**)&d_a, size);
    hipMalloc((void**)&d_b, size);
    hipMalloc((void**)&d_c, size);

    /* Copy data from host to device */
    hipMemcpy(d_a, a, size, hipMemcpyHostToDevice);
    hipMemcpy(d_b, b, size, hipMemcpyHostToDevice);

    /* 
    Kernel Launch
    Grid contains M*N blocks
    Each block has 1 thread
    Hence index of matrix element is
    blockIdx.x* gridSize.y + blockIdx.y
    */
    dim3 gridSize(M, N);
    matAdd<<<gridSize, 1>>>(d_a, d_b, d_c);
    hipDeviceSynchronize();

    /* Copy result from GPU device to host */
    hipMemcpy(c, d_c, size, hipMemcpyDeviceToHost);
    
    /* Print result */
    std::cout<<"A + B =\n";
    print_matrix(c);
    std::cout<<'\n';

    /* Cleanup device and host memory */
    hipFree(d_a);
    hipFree(d_b);
    hipFree(d_c);
    delete a;
    delete b;
    delete c;

    return 0;
}
